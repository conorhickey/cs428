#include "hip/hip_runtime.h"
#include "image.h"
#include "sepia.h"

/* kernel function */
/*   inbuf:  pixels of the original image */
/*   outbuf: pixels of the sepia image */
/*   r:      number of rows */
/*   c:      number of cols */
__global__ sepia_ker(struct pixel *inbuf, struct pixel *outbuf, int r, int c)
{
    /* Your code goes here */
}

/* in:  original image */
/* out: sepia image    */
void sepia(struct image *im, struct im *sim)
{
    /* Your code goes here */
}
