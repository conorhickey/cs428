#include "hip/hip_runtime.h"
//allows the program to compile as C
extern "C"{
#include "sepia.h"
}
#include "image.h"
#include "sepia.h"
#include <stdlib.h>

/* kernel function */
/*   inbuf:  pixels of the original image */
/*   outbuf: pixels of the sepia image */
/*   r:      number of rows */
/*   c:      number of cols */
__global__ void sepia_ker(struct pixel *inbuf, struct pixel *outbuf, int r, int c)
{	

	//Code similar to sequential code excpet it uses a value i calucalted
	//from 	i = (blockIdx.x * blockDim.x) * threadIdx.x;
	//inbuf and outbuf used instead of PixP and simPixP

	int i;
	uint32_t temp;
	i = (blockIdx.x * blockDim.x) * threadIdx.x;

	if(i<r*c){ 
		temp = 0.393f * inbuf[i].r + 0.769f * inbuf[i].g + 0.189f * inbuf[i].b;    	
	   	if(temp > 65535){
	   	outbuf[i].r = 65535;
	   	}
	   	else{
	   	     outbuf[i].r =temp;
		}

		temp = 0.349f * inbuf[i].r + 0.686f * inbuf[i].g + 0.168f * inbuf[i].b;
		if(temp > 65535){
		outbuf[i].g = 65535;
		}
		else{
			outbuf[i].g = temp;
		}
			
		temp = 0.272f * inbuf[i].r + 0.534f * inbuf[i].g + 0.131f * inbuf[i].b;
		if(temp > 65535){
		outbuf[i].b = 65535;
		}
		else{
			outbuf[i].b = temp;
		}
		
		outbuf[i].a = inbuf[i].a;

		
		
     }
}


/* in:  original image */
/* out: sepia image    */
void sepia(struct image *im, struct image *sim)
{	
	//sets up various values and pixels
	sim->rows= im->rows;
    sim->cols= im->cols;
    struct pixel *dpixSimP,*dpixP;
    struct pixel *pixSimP,*pixP;   
	
	
	//used to calculate the threads and block size
    int n = sim->rows * sim->cols;
    int size = n*sizeof(struct pixel);
    int n_threads = 1024;
    int n_blocks = (n-1)/(n_threads +1);
	printf("%d\n",n_blocks);
    sim->pixbuf = (struct pixel *)malloc(size);
    pixP = im->pixbuf;
	pixSimP = sim->pixbuf;
	
	//allocates the cuda memory
	hipMalloc((void **) &dpixSimP,size);
	hipMalloc((void **) &dpixP,size);
	
	//copies it to device
	hipMemcpy(dpixP,im->pixbuf,size,hipMemcpyHostToDevice);
	sepia_ker<<<n_blocks,n_threads>>>(im->pixbuf,sim->pixbuf,im->rows,im->cols);
	
	//copies output from device to host
	hipMemcpy(sim->pixbuf, dpixSimP, size,hipMemcpyDeviceToHost);
	
	//Frees up the memory 
	hipFree(dpixSimP);
	hipFree(dpixP);
	

}
