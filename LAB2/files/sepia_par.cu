#include "hip/hip_runtime.h"

#include "image.h"
#include "sepia.h"
#include <stdlib.h>

/* kernel function */
/*   inbuf:  pixels of the original image */
/*   outbuf: pixels of the sepia image */
/*   r:      number of rows */
/*   c:      number of cols */
__global__ void sepia_ker(struct pixel *inbuf, struct pixel *outbuf, int r, int c)
{
	int i;
	uint32_t temp;
	i = blockIdx.x * blockDim.x * threadIdx.x;
	for(i=0;i<r*c;i++){
	//if(i<r*c){    	
		temp = 0.393f * inbuf[i].r + 0.769f * inbuf[i].g + 0.189f * inbuf[i].b;    	
	   	if(temp > 65535){
	   	outbuf[i].r = 65535;
	   	}
	   	else{
	   	     outbuf[i].r =temp;
		}

		temp = 0.349f * inbuf[i].r + 0.686f * inbuf[i].g + 0.168f * inbuf[i].b;
		if(temp > 65535){
		outbuf[i].g = 65535;
		}
		else{
			outbuf[i].g = temp;
		}
			
		temp = 0.272f * inbuf[i].r + 0.534f * inbuf[i].g + 0.131f * inbuf[i].b;
		if(temp > 65535){
		outbuf[i].b = 65535;
		}
		else{
			outbuf[i].b = temp;
		}
		
		outbuf[i].a = inbuf[i].a;
		
     }
}


/* in:  original image */
/* out: sepia image    */
void sepia(struct image *im, struct image *sim)
{	
	sim->rows= im->rows;
    sim->cols= im->cols;
    struct pixel *dpixSimP,*dpixP;
    struct pixel *pixSimP,*pixP;
    int n = sim->rows * sim->cols;
    int n_threads = 1024;
    int n_blocks = (n-1)/1024 +1;
	printf("%d\n",n_blocks);
    sim->pixbuf = (struct pixel *)malloc(n*sizeof(struct pixel));
    pixP = im->pixbuf;
	pixSimP = sim->pixbuf;
	hipMalloc((void **) &dpixSimP,n*sizeof(struct pixel));
	hipMalloc((void **) &dpixP,n*sizeof(struct pixel));
	
	hipMemcpy(dpixSimP,pixSimP, n*sizeof(struct pixel),hipMemcpyHostToDevice);
	hipMemcpy(&dpixP, pixP, n*sizeof(struct pixel),hipMemcpyHostToDevice);
	sepia_ker<<<n_blocks,n_threads>>>(sim->pixbuf,im->pixbuf,im->rows,im->cols);
	
	
	hipFree(dpixSimP);
	hipFree(dpixP);
	

}
